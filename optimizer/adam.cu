#include "adam.h"
void initAdam(Adam *optimizer, NN *model, float lr, int clip)
{
    optimizer->model = model;
    optimizer->lr = lr;
    optimizer->clip = clip;
    optimizer->moments = (Moments *)malloc(sizeof(Moments) * model->num_of_layers);

    for (int i = 0; i < optimizer->model->num_of_layers; i++)
    {
        optimizer->moments[i].moment1_W = createMatrix(model->layers[i].weights->rows, model->layers[i].weights->columns);
        optimizer->moments[i].moment1_b = createMatrix(model->layers[i].biases->rows, model->layers[i].biases->columns);
        optimizer->moments[i].moment2_W = createMatrix(model->layers[i].weights->rows, model->layers[i].weights->columns);
        optimizer->moments[i].moment2_b = createMatrix(model->layers[i].biases->rows, model->layers[i].biases->columns);
    }
}
void AdamOptimizer(Adam *optimizer)
{
    for (int i = 0; i < optimizer->model->num_of_layers; i++)
    {
        int size = optimizer->moments[i].moment1_W->rows * optimizer->moments[i].moment1_W->columns;
        int nBlocks = (size - 1) / BlockSize + 1;
        AdamOptimizerKernel<<<nBlocks, BlockSize>>>(optimizer->model->layers[i].weights->data, optimizer->model->layers[i].weight_gradients->data,
                                                   optimizer->moments[i].moment1_W->data, optimizer->moments[i].moment2_W->data, size, optimizer->lr, beta1, beta2, epsilon, optimizer->clip);
        hipDeviceSynchronize();
        size = optimizer->moments[i].moment1_b->rows * optimizer->moments[i].moment1_b->columns;
        nBlocks = (size - 1) / BlockSize + 1;
        AdamOptimizerKernel<<<nBlocks, BlockSize>>>(optimizer->model->layers[i].biases->data, optimizer->model->layers[i].bias_gradients->data,
                                                   optimizer->moments[i].moment1_b->data, optimizer->moments[i].moment2_b->data, size, optimizer->lr, beta1, beta2, epsilon, optimizer->clip);
        hipDeviceSynchronize();
    }
}
void freeAdam(Adam *optimizer)
{
    for (int i = 0; i < optimizer->model->num_of_layers; i++)
    {
        freeMatrix(optimizer->moments[i].moment1_W);
        freeMatrix(optimizer->moments[i].moment1_b);
        freeMatrix(optimizer->moments[i].moment2_W);
        freeMatrix(optimizer->moments[i].moment2_b);
    }
    free(optimizer->moments);
}